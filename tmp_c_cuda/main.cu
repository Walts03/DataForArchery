
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to add the elements of two arrays
__global__
void Add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
  }
}

int main(void) {
  int n = 1 << 20;  // 1M elements

  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, n * sizeof(float));
  hipMallocManaged(&y, n * sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < n; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int block_size = 256;
  int num_blocks = (n + block_size - 1) / block_size;
  Add<<<num_blocks, block_size>>>(n, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float max_error = 0.0f;
  for (int i = 0; i < n; i++) {
    max_error = fmax(max_error, fabs(y[i] - 3.0f));
  }
  printf("Max error: %f\n", max_error);

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}

